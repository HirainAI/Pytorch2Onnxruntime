#include "hip/hip_runtime.h"
/*
batch version of point sampling and gathering, modified from the original implementation of official PointNet++ codes.
Written by Shaoshuai Shi
All Rights Reserved 2018.
*/


#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "sampling_gpu.h"

__global__ void gather_points_kernel_fast(int b, int c, int n, int m, 
    const float *__restrict__ points, const int *__restrict__ idx, float *__restrict__ out) {
    // points: (B, C, N)
    // idx: (B, M)
    // output:
    //      out: (B, C, M)

    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || c_idx >= c || pt_idx >= m) return;

    out += bs_idx * c * m + c_idx * m + pt_idx;
    idx += bs_idx * m + pt_idx;
    points += bs_idx * c * n + c_idx * n;
    out[0] = points[idx[0]];
}

void gather_points_kernel_launcher_fast(int b, int c, int n, int npoints, 
    const float *points, const int *idx, float *out) {
    // points: (B, C, N)
    // idx: (B, npoints)
    // output:
    //      out: (B, C, npoints)

    hipError_t err;
    dim3 blocks(DIVUP(npoints, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    gather_points_kernel_fast<<<blocks, threads>>>(b, c, n, npoints, points, idx, out);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

__global__ void gather_points_grad_kernel_fast(int b, int c, int n, int m, const float *__restrict__ grad_out, 
    const int *__restrict__ idx, float *__restrict__ grad_points) {
    // grad_out: (B, C, M)
    // idx: (B, M)
    // output:
    //      grad_points: (B, C, N)

    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || c_idx >= c || pt_idx >= m) return;

    grad_out += bs_idx * c * m + c_idx * m + pt_idx;
    idx += bs_idx * m + pt_idx;
    grad_points += bs_idx * c * n + c_idx * n;

    atomicAdd(grad_points + idx[0], grad_out[0]);
}

void gather_points_grad_kernel_launcher_fast(int b, int c, int n, int npoints, 
    const float *grad_out, const int *idx, float *grad_points) {
    // grad_out: (B, C, npoints)
    // idx: (B, npoints)
    // output:
    //      grad_points: (B, C, N)

    hipError_t err;
    dim3 blocks(DIVUP(npoints, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    gather_points_grad_kernel_fast<<<blocks, threads>>>(b, c, n, npoints, grad_out, idx, grad_points);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__device__ void __update(float *__restrict__ dists, int *__restrict__ dists_i, int idx1, int idx2){
    const float v1 = dists[idx1], v2 = dists[idx2];
    const int i1 = dists_i[idx1], i2 = dists_i[idx2];
    dists[idx1] = max(v1, v2);
    dists_i[idx1] = v2 > v1 ? i2 : i1;
}

template <unsigned int block_size>
__global__ void furthest_point_sampling_kernel(int b, int n, int m, 
    const float *__restrict__ dataset, float *__restrict__ temp, int *__restrict__ idxs) {
    // dataset: (B, N, 3)
    // tmp: (B, N)
    // output:
    //      idx: (B, M)
    printf("-----------------------");
    printf("idxs1 is %d\n",m);
    
    printf("dataset is %f\n",dataset[0]);
    printf("temp is %f\n",temp[0]);
    if (m <= 0) return;
    // shared memory can be accessed by all the threads inside the block
    __shared__ float dists[block_size];     // store distances
    __shared__ int dists_i[block_size];     // index?
    printf("idxs2 is %d\n",m);
    int batch_index = blockIdx.x;           // seems that blockIdx.x means the frame idx inside the batch
    dataset += batch_index * n * 3;         // ptr offset, now the ptr dataset point to the first element in the frame
    temp += batch_index * n;
    idxs += batch_index * m;

    int tid = threadIdx.x;                  // tid in (1024/512/256...)
    const int stride = block_size;          // num of threads, may contains 1 / multiple points to be sampled from
    printf("idxs3 is %d\n",threadIdx.x);
    int old = 0;
    if (threadIdx.x == 0)
    idxs[0] = old;                          // the first point to be sampled -> picked the first point in n directly
    printf("idxs4 is %d\n",m);
    printf("idxs4 is %d\n",m);
    // synchronizing... as shared memory accessed by all the threads in the block, make idxs[0] visible to all threads?
    __syncthreads();
    printf("idxs5 is %d\n",m);
    for (int j = 1; j < m; j++) {           // the first point already picked (seed point), loop m-1 to be sampled pts
    printf("idxs6 is %d\n",idxs[j]);
    int besti = 0;                          // best index
    float best = -1;                        // best distance?
    float x1 = dataset[old * 3 + 0];        // get the coordinates of current idx point
    float y1 = dataset[old * 3 + 1];
    float z1 = dataset[old * 3 + 2];
    for (int k = tid; k < n; k += stride) {     // start from first source point in the thread
        float x2, y2, z2;
        x2 = dataset[k * 3 + 0];                // get the coordinates of the k+stride idx point
        y2 = dataset[k * 3 + 1];
        z2 = dataset[k * 3 + 2];
        // float mag = (x2 * x2) + (y2 * y2) + (z2 * z2);
        // if (mag <= 1e-3)
        // continue;

        // TODO try to store the d
        float d = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
        float d2 = min(d, temp[k]);             // the min distance of other points to the k-th point
        temp[k] = d2;                           // update the min distance of the k-th point
        besti = d2 > best ? k : besti;
        best = d2 > best ? d2 : best;
    }
    dists[tid] = best;
    dists_i[tid] = besti;
    __syncthreads();

    if (block_size >= 1024) {
        if (tid < 512) {
            __update(dists, dists_i, tid, tid + 512);
        }
        __syncthreads();
    }

    if (block_size >= 512) {
        if (tid < 256) {
            __update(dists, dists_i, tid, tid + 256);
        }
        __syncthreads();
    }
    if (block_size >= 256) {
        if (tid < 128) {
            __update(dists, dists_i, tid, tid + 128);
        }
        __syncthreads();
    }
    if (block_size >= 128) {
        if (tid < 64) {
            __update(dists, dists_i, tid, tid + 64);
        }
        __syncthreads();
    }
    if (block_size >= 64) {
        if (tid < 32) {
            __update(dists, dists_i, tid, tid + 32);
        }
        __syncthreads();
    }
    if (block_size >= 32) {
        if (tid < 16) {
            __update(dists, dists_i, tid, tid + 16);
        }
        __syncthreads();
    }
    if (block_size >= 16) {
        if (tid < 8) {
            __update(dists, dists_i, tid, tid + 8);
        }
        __syncthreads();
    }
    if (block_size >= 8) {
        if (tid < 4) {
            __update(dists, dists_i, tid, tid + 4);
        }
        __syncthreads();
    }
    if (block_size >= 4) {
        if (tid < 2) {
            __update(dists, dists_i, tid, tid + 2);
        }
        __syncthreads();
    }
    if (block_size >= 2) {
        if (tid < 1) {
            __update(dists, dists_i, tid, tid + 1);
        }
        __syncthreads();
    }

    old = dists_i[0];
    if (tid == 0)
        idxs[j] = old;
    }
    
}

void furthest_point_sampling_kernel_launcher(int64_t b, int64_t n, int64_t m, 
    const float *dataset, float *temp, int64_t *idxs) {
    // dataset: (b, n, 3) float
    // tmp: (b, n) float
    // output:
    //      idx: (b, m)
    printf("=======cuda");
    hipError_t err;
    unsigned int n_threads = opt_n_threads(n);    // get the optimal num of threads? 1-dim structure

    // set the block size in template
    // grid size equal to batch size
    // block size equal to n_threads according to n input num of points
    switch (n_threads) {
        case 1024:
        printf("=======1");
        furthest_point_sampling_kernel<1024><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 512:
        printf("=======2");
        furthest_point_sampling_kernel<512><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 256:
        printf("=======3");
        furthest_point_sampling_kernel<256><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 128:
        printf("=======4");
        furthest_point_sampling_kernel<128><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 64:
        printf("=======5");
        furthest_point_sampling_kernel<64><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 32:
        printf("=======6");
        furthest_point_sampling_kernel<32><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 16:
        printf("=======7");
        furthest_point_sampling_kernel<16><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 8:
        printf("=======9");
        furthest_point_sampling_kernel<8><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 4:
        printf("=======10");
        furthest_point_sampling_kernel<4><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 2:
        printf("=======8");
        furthest_point_sampling_kernel<2><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        case 1:
        printf("=======11");
        furthest_point_sampling_kernel<1><<<b, n_threads>>>(b, n, m, dataset, temp, idxs); break;
        default:
        printf("=======12");
        furthest_point_sampling_kernel<512><<<b, n_threads>>>(b, n, m, dataset, temp, idxs);
    }

    err = hipGetLastError();
    if (hipSuccess != err) {
        printf("=======22");
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


template <unsigned int block_size>
__global__ void furthest_point_sampling_with_dist_kernel(int b, int n, int m,
    const float *__restrict__ dataset, float *__restrict__ temp, int *__restrict__ idxs) {
    // dataset: (B, N, N)
    // tmp: (B, N)
    // output:
    //      idx: (B, M)

    if (m <= 0) return;
    __shared__ float dists[block_size];
    __shared__ int dists_i[block_size];

    int batch_index = blockIdx.x;
    dataset += batch_index * n * n;
    temp += batch_index * n;
    idxs += batch_index * m;

    int tid = threadIdx.x;
    const int stride = block_size;

    int old = 0;
    if (threadIdx.x == 0)
    idxs[0] = old;

    __syncthreads();
    for (int j = 1; j < m; j++) {
    int besti = 0;
    float best = -1;
    // float x1 = dataset[old * 3 + 0];
    // float y1 = dataset[old * 3 + 1];
    // float z1 = dataset[old * 3 + 2];
    for (int k = tid; k < n; k += stride) {
        // float x2, y2, z2;
        // x2 = dataset[k * 3 + 0];
        // y2 = dataset[k * 3 + 1];
        // z2 = dataset[k * 3 + 2];

        // float d = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
        float d = dataset[old * n + k];

        float d2 = min(d, temp[k]);
        temp[k] = d2;
        besti = d2 > best ? k : besti;
        best = d2 > best ? d2 : best;
    }
    dists[tid] = best;
    dists_i[tid] = besti;
    __syncthreads();

    if (block_size >= 1024) {
        if (tid < 512) {
            __update(dists, dists_i, tid, tid + 512);
        }
        __syncthreads();
    }

    if (block_size >= 512) {
        if (tid < 256) {
            __update(dists, dists_i, tid, tid + 256);
        }
        __syncthreads();
    }
    if (block_size >= 256) {
        if (tid < 128) {
            __update(dists, dists_i, tid, tid + 128);
        }
        __syncthreads();
    }
    if (block_size >= 128) {
        if (tid < 64) {
            __update(dists, dists_i, tid, tid + 64);
        }
        __syncthreads();
    }
    if (block_size >= 64) {
        if (tid < 32) {
            __update(dists, dists_i, tid, tid + 32);
        }
        __syncthreads();
    }
    if (block_size >= 32) {
        if (tid < 16) {
            __update(dists, dists_i, tid, tid + 16);
        }
        __syncthreads();
    }
    if (block_size >= 16) {
        if (tid < 8) {
            __update(dists, dists_i, tid, tid + 8);
        }
        __syncthreads();
    }
    if (block_size >= 8) {
        if (tid < 4) {
            __update(dists, dists_i, tid, tid + 4);
        }
        __syncthreads();
    }
    if (block_size >= 4) {
        if (tid < 2) {
            __update(dists, dists_i, tid, tid + 2);
        }
        __syncthreads();
    }
    if (block_size >= 2) {
        if (tid < 1) {
            __update(dists, dists_i, tid, tid + 1);
        }
        __syncthreads();
    }

    old = dists_i[0];
    if (tid == 0)
        idxs[j] = old;
    }
}


void furthest_point_sampling_with_dist_kernel_launcher(int b, int n, int m,
    const float *dataset, float *temp, int *idxs, hipStream_t stream) {
    // dataset: (B, N, N)
    // tmp: (B, N)
    // output:
    //      idx: (B, M)

    hipError_t err;
    unsigned int n_threads = opt_n_threads(n);

    switch (n_threads) {
        case 1024:
        furthest_point_sampling_with_dist_kernel<1024><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 512:
        furthest_point_sampling_with_dist_kernel<512><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 256:
        furthest_point_sampling_with_dist_kernel<256><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 128:
        furthest_point_sampling_with_dist_kernel<128><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 64:
        furthest_point_sampling_with_dist_kernel<64><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 32:
        furthest_point_sampling_with_dist_kernel<32><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 16:
        furthest_point_sampling_with_dist_kernel<16><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 8:
        furthest_point_sampling_with_dist_kernel<8><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 4:
        furthest_point_sampling_with_dist_kernel<4><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 2:
        furthest_point_sampling_with_dist_kernel<2><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        case 1:
        furthest_point_sampling_with_dist_kernel<1><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs); break;
        default:
        furthest_point_sampling_with_dist_kernel<512><<<b, n_threads, 0, stream>>>(b, n, m, dataset, temp, idxs);
    }

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
